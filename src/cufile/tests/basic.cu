/*
 * SPDX-FileCopyrightText: 2025 Ben Jarvis
 * SPDX-License-Identifier: Unlicense
 */

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <fcntl.h>
#include <errno.h>
#include <unistd.h>

#include <hip/hip_runtime.h>
#include <cufile.h>

extern "C"
{
#include "client/client.h"
#include "common/logging.h"
#include "cufile/chimera_cufile.h"
};

int main(int argc, char **argv)
{
    int fd;
    ssize_t ret;
    void *devPtr_base;
    off_t file_offset = 0x2000;
    off_t devPtr_offset = 0x1000;
    ssize_t IO_size = 1UL << 24;
    size_t buff_size = IO_size + 0x1000;
    CUfileError_t status;
    // CUResult cuda_result;
    int cuda_result;
    CUfileDescr_t cf_descr;
    CUfileHandle_t cf_handle;
    char *testfn;

    chimera_log_init();

    ChimeraLogLevel = CHIMERA_LOG_DEBUG;

    if (argc < 2)
    {
        fprintf(stderr, "Usage: %s <testfile>\n", argv[0]);
        return -1;
    }

    testfn = argv[1];

    fprintf(stderr, "Opening File %s\n", testfn);

    fd = open(testfn, O_CREAT | O_WRONLY | O_DIRECT, 0644);
    if (fd < 0)
    {
        fprintf(stderr, "file open %s errno %d\n", testfn, errno);
        return -1;
    }

    // the above fd could also have been opened without O_DIRECT starting CUDA toolkit 12.2
    // (gds 1.7.x version) as follows
    // fd = open(testfn, O_CREAT|O_WRONLY, 0644);

    fprintf(stderr, "Opening cuFileDriver.\n");
    status = cuFileDriverOpen();
    if (status.err != CU_FILE_SUCCESS)
    {
        fprintf(stderr, " cuFile driver failed to open\n");
        close(fd);
        return -1;
    }

    fprintf(stderr, "Registering cuFile handle to %s.\n", testfn);

    fprintf(stderr, "chimera_cudesc_init\n");
    chimera_cudesc_init(&cf_descr, &cf_descr);
    /* XXX */
    // cf_descr.handle.fd = fd;

    status = cuFileHandleRegister(&cf_handle, &cf_descr);

    if (status.err != CU_FILE_SUCCESS)
    {
        fprintf(stderr, "cuFileHandleRegister fd %d status %d\n", fd, status.err);
        close(fd);
        return -1;
    }

    fprintf(stderr, "Allocating CUDA buffer of %zu bytes.\n", buff_size);

    cuda_result = hipMalloc(&devPtr_base, buff_size);
    if (cuda_result != hipSuccess)
    {
        fprintf(stderr, "buffer allocation failed %d\n", cuda_result);
        cuFileHandleDeregister(cf_handle);
        close(fd);
        return -1;
    }

    fprintf(stderr, "Registering Buffer of %zu bytes.\n", buff_size);
    status = cuFileBufRegister(devPtr_base, buff_size, 0);
    if (status.err != CU_FILE_SUCCESS)
    {
        fprintf(stderr, "buffer registration failed %d\n", status.err);
        cuFileHandleDeregister(cf_handle);
        close(fd);
        hipFree(devPtr_base);
        return -1;
    }

    // fill a pattern
    fprintf(stderr, "Filling memory.\n");

    hipMemset((void *)devPtr_base, 0xab, buff_size);
    hipStreamSynchronize(0);

    // perform write operation directly from GPU mem to file
    fprintf(stderr, "Writing buffer to file.\n");
    ret = cuFileWrite(cf_handle, devPtr_base, IO_size, file_offset, devPtr_offset);

    if (ret < 0 || ret != IO_size)
    {
        fprintf(stderr, "cuFileWrite failed %ld\n", ret);
    }

    // release the GPU memory pinning
    fprintf(stderr, "Releasing cuFile buffer.\n");
    status = cuFileBufDeregister(devPtr_base);
    if (status.err != CU_FILE_SUCCESS)
    {
        fprintf(stderr, "buffer deregister failed\n");
        hipFree(devPtr_base);
        cuFileHandleDeregister(cf_handle);
        close(fd);
        return -1;
    }

    fprintf(stderr, "Freeing CUDA buffer.\n");
    hipFree(devPtr_base);
    // deregister the handle from cuFile
    fprintf(stderr, "Releasing file handle. \n");
    (void)cuFileHandleDeregister(cf_handle);
    close(fd);

    // release all cuFile resources
    fprintf(stderr, "Closing File Driver.\n");
    (void)cuFileDriverClose();

    fprintf(stderr, "\n");

    return 0;
}
